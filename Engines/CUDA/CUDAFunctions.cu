#include "hip/hip_runtime.h"
/* Trying to implement the following algorithm using CUDA */
//for(int w = 0; w<frame->frame->getWidth(); w++){
//    for(int h = 0; h<frame->frame->getHeight(); h++){
//        for(int v = 0; v<verticesInPixel[w][h].size(); v++){

//            if(!(verticesInPixel[w][h][v]->assigned)){
//                // This is the case where the vertex has never been assigned a value before
//                if(verticesInPixel[w][h][v]->zDepth>1.05*depthPixels[w][h]){
//                    verticesInPixel[w][h][v]->TexCoords = glm::vec2(0,0);
//                } else {
//                    verticesInPixel[w][h][v]->TexCoords = glm::vec2(verticesInPixel[w][h][v]->TexCoords.x/640, verticesInPixel[w][h][v]->TexCoords.y/(480*totalFr)+((float)frNr/(float)totalFr));
//                    verticesInPixel[w][h][v]->assigned = true;
//                }
//            }
//        }
//    }
//}


/*
 * inputs: std::vector<std::vector<std::vector<*Vertex>>>
 * outputs: Modified
*/
#include <CUDAFunctions.h>
void findTexCoord(std::vector<std::vector<float> > *depthPixels,std::vector<std::vector<std::vector<Vertex *> > > *PixelsInImage, int* fr, int*frNr){
    int width = depthPixels->size();
    int height = depthPixels[0].size();
    findTexCoords<<<height,width>>>(depthPixels, PixelsInImage, fr, frNr);
}

__global__ void findTexCoords(std::vector<std::vector<float> > *depthPixels, std::vector<std::vector<std::vector<Vertex *> > > *verticesInPixel, int *fr, int *frNr){
    int w = blockIdx.x;
    int h = threadIdx.x;
    int FrameNumber = *fr; int TotalFrames = *frNr;
    for(int v = 0; v<(*verticesInPixel)[w][h].size(); v++){
        if(!((*verticesInPixel)[w][h][v]->assigned)){
            // This is the case where the vertex has never been assigned a value before
            if((*verticesInPixel)[w][h][v]->zDepth>1.05*(*depthPixels)[w][h]){
                (*verticesInPixel)[w][h][v]->TexCoords = glm::vec2(0,0);
            } else {
                (*verticesInPixel)[w][h][v]->TexCoords = glm::vec2((*verticesInPixel)[w][h][v]->TexCoords.x/640, (*verticesInPixel)[w][h][v]->TexCoords.y/(480*TotalFrames)+((float)FrameNumber/(float)TotalFrames));
                (*verticesInPixel)[w][h][v]->assigned = true;
            }
        }
    }

}
